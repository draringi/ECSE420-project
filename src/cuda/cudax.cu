#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
extern "C" {
#include <stdlib.h>
#include <stdio.h>
#include <limits.h>
#include "cudax.h"
}

__device__ void cardinalities_calculations (int* cardinalities, int* min_cardinality, struct MATRIX_ENTRY** cols, int col_count){
  int i;
  for (i=0; i < col_count; i++){
    if(cardinalities[i]==-1){
      continue;
    }
    cardinalities[i]=0;
    struct MATRIX_ENTRY* e = cols[i];
    if(e){
      do {
        cardinalities[i]++;
        e = e->down;
      } while(e!=cols[i]);
    }
    if(cardinalities[i]< (*min_cardinality)){
      *min_cardinality = cardinalities[i];
    }
  }
}

__global__ void solve_cover ( struct MATRIX_ENTRY* entries, int* results , size_t list_size, uint32_t width, uint32_t height) {
  size_t results_base = blockIdx.x*height;
  size_t results_end = results_base + height;
  size_t entry_base = blockIdx.x*list_size;
  size_t entry_end = entry_base + list_size;
  size_t i;
  struct MATRIX_ENTRY** cols = (struct MATRIX_ENTRY**) malloc(width * sizeof(struct MATRIX_ENTRY*));
  for(i = 0; i < width; i++){
    cols[i] = NULL;
  }
  struct MATRIX_ENTRY** rows = (struct MATRIX_ENTRY**) malloc(height * sizeof(struct MATRIX_ENTRY*));
  for(i=0; i < height; i++){
    rows[i] = NULL;
  }
  // Find start of each row and column.
  for(i=entry_base; i<entry_end; i++){
    int col_id = entries[i].x;
    int row_id = entries[i].y;
    if(cols[col_id]){
      if(cols[col_id]->y>row_id){
        cols[col_id] = &entries[i];
      }
    } else {
      cols[col_id] = &entries[i];
    }
    if(rows[row_id]){
      if(rows[row_id]->x>col_id){
        rows[row_id] = &entries[i];
      }
    } else {
      rows[row_id] = &entries[i];
    }
  }
  //link
  for(i=entry_base; i<entry_end; i++){
    struct MATRIX_ENTRY* e = &entries[i];
		struct MATRIX_ENTRY* r = rows[e->y];
		if(r!=e){
			while(r->right&&r->right->x < e->x){
				r = r->right;
			}
			if(r->right){
				e->right = r->right;
				e->right->left = e;
			}
			e->left = r;
			r->right = e;
		}
		struct MATRIX_ENTRY* c = cols[e->x];
		if(c!=e){
			while(c->down&&c->down->y < e->y){
				c = c->down;
			}
			if (c->down) {
				e->down = c->down;
				e->down->up = e;
			}
			e->up = c;
			c->down = e;
		}
  }
  for(i = 0; i < height; i ++){
    struct MATRIX_ENTRY* front = rows[i];
    struct MATRIX_ENTRY* end = front;
    while(end->right){
      end = end->right;
    }
    end->right = front;
    front->left = end;
  }
  for(i = 0; i < width; i++){
    struct MATRIX_ENTRY* top = cols[i];
    struct MATRIX_ENTRY* bottom = top;
    while(bottom->down){
      bottom = bottom->down;
    }
    bottom->down = top;
    top->up = bottom;
  }
  for(i = results_base; i < results_end; i++){
    results[i] = -1;
  }
  size_t result_count = 0;
  size_t columns_left = width;
  size_t rows_left = height;
  int* cardinalities = (int*) malloc(width*sizeof(int));
  int min_cardinality = INT_MAX;
  cardinalities_calculations(cardinalities, &min_cardinality, cols, width);
  //Solve;
  while(columns_left){
    if(!rows_left||min_cardinality==0){
      results[0] = -1;
      break;
    }
    i = 0;
    while (cardinalities[i] != min_cardinality){
      i++;
    }
    size_t to_choose = blockIdx.x % (min_cardinality+1);
    size_t row_count;
    struct MATRIX_ENTRY* e = cols[i];
    for (row_count = 0; row_count < to_choose; row_count++){
      e = e->down;
    }
    struct MATRIX_ENTRY* r = rows[e->y];
    do{
      struct MATRIX_ENTRY* c = r->down;
      while(r != c){
        struct MATRIX_ENTRY* rem = c->right;
        while (rem != c){
          rem->up->down = rem->down;
          rem->down->up = rem->up;
          if(rem == cols[rem->x]){
            if(rem->down == rem){
              cols[rem->x] = NULL;
            } else {
              cols[rem->x] = rem->down;
            }
          }
          rem = rem->right;
        }
        rows[c->y] = NULL;
        c = c->down;
        --rows_left;
      }
      cols[r->x] = NULL;
      cardinalities[r->x] = -1;
      r = r->right;
      --columns_left;
    } while (r != rows[e->y]);
    --rows_left;
    rows[e->y] = NULL;
    cardinalities_calculations(cardinalities, &min_cardinality, cols, width);
    results[results_base+result_count] = e->y;
    ++result_count;
  }
  free(cols);
  free(cardinalities);
  free(rows);
}

uint64_t factorial(uint32_t n){
  uint64_t value = 1;
  while (n > 1){
    value *= n;
    --n;
  }
  return n;
}

void find_cover(struct MATRIX_ENTRY* entries_list, size_t list_size, uint32_t width, uint32_t height){
  uint64_t* cardinalities = (uint64_t*)calloc(width, sizeof(uint64_t));
  size_t i;
  for(i=0; i<list_size; i++){
    int col = entries_list[i].x;
    cardinalities[col]++;
  }
  uint64_t cardinality_product = 1;
  for(i=0; i<list_size; i++){
    cardinality_product *= cardinalities[i];
  }
  uint64_t row_factorial = factorial(height);
  uint64_t cores = (cardinality_product > row_factorial) ? row_factorial : cardinality_product;
  void* data_array;
  size_t len = list_size*sizeof(struct MATRIX_ENTRY);
  size_t array_size = cores*len;
  hipError_t err = hipMalloc(&data_array, array_size);
  if(err != hipSuccess){
    abort();
  }
  for(i = 0; i < cores; i++){
    void* ptr = (void*)(data_array + (i*len));
    err = hipMemcpy(ptr, entries_list, len, hipMemcpyHostToDevice);
    if(err != hipSuccess){
      abort();
    }
  }
  size_t results_size = height*cores*sizeof(int);
  void* results;
  err = hipMalloc(&results, results_size);
  if(err != hipSuccess){
    abort();
  }
  solve_cover<<<cores, 1>>>((struct MATRIX_ENTRY*)data_array, (int*)results, list_size, width, height);
  int* local_results = (int*) malloc(results_size);
  err = hipMemcpy(results, local_results, results_size, hipMemcpyDeviceToHost);
  if(err != hipSuccess){
    abort();
  }
  size_t success_count = 0;
  for (i=0; i<cores; i++){
    size_t res_length = 0;
    size_t j;
    size_t base = i*height;
    for(j = base; j < base + height; j++){
      int val = local_results[j];
      if(val == -1) {
        break;
      }
      printf("%d\t", val);
      res_length++;
    }
    if(res_length>0){
      printf("\n");
      success_count++;
    }
  }
  if(success_count == 0){
    printf("No Solutions Found\n");
  }
}
